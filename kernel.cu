#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#define N 10
#define M 10

using namespace std;

hipError_t calculatePrimes(int *c, int *a, unsigned int size);

hipError_t transposeMatrix(int (&matrix_a)[N][M], int (&t_pose)[N][M]);

__global__ void transposeKernal(int *matrix_a, int *transpose) {
	int i = threadIdx.x;
	int x = blockDim.x;
	int j = blockIdx.x;

	transpose[x * j + i] = matrix_a[x * i + j];
}

__global__ void primeKernel(int *c, int *a){
	int i = threadIdx.x;
	if (i % 2 != 0) {
		c[i] = (6 * a[i]) + 1;
		c[i + 1] = (6 * a[i] - 1);
	}
}

int main(){
	//const int arraySize = 50;
	/*int *num_holder = (int*)malloc(sizeof(int) * arraySize);
	int *prime_holder = (int*)malloc(sizeof(int) * arraySize);*/
	//prime_holder[arraySize + 1] += '\0';

	//insert numbers to a
	/*for (int i = 0; i < arraySize; i++) {
		num_holder[i] = i;
	}*/

	// Add vectors in parallel.
	//hipError_t cudaStatus = calculatePrimes(prime_holder, num_holder, arraySize);
	/*if (cudaStatus != hipSuccess) {
		fprintf(stderr, "calculatePrimes failed!");
		return 1;
	}*/

	/*for (int i = 0; i < arraySize; i++) {
		printf("6(%d) + 1 = %d\n", num_holder[i], prime_holder[i]);
	}*/
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
   /* cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}*/

	size_t size = (N * M) * sizeof(int);
	
	hipError_t mainStatus;
	
	int matrix_a[N][M];
	int matrix_b[N][M];
	int matrix_c[N][M];
	
	for(int i = 0; i < N; i++) {
		for(int j = 0; j < M; j++) {
			matrix_a[i][j] = 1 + rand() / 100;
		}
	}

	mainStatus = transposeMatrix(matrix_a, matrix_b);
	if(mainStatus != hipSuccess) {
		fprintf(stderr, "failed in main\n");
	}

	cout << "Before Transpose" << endl;
	for(int i = 0; i < N; i++) {
		for(int j = 0; j < M; j++) {
			cout << matrix_a[i][j] << " ";
		}
		cout << endl;
	}


	cout << "After Transpose" << endl;
	for(int i = 0; i < N; i++) {
		for(int j = 0; j < M; j++) {
			cout << matrix_b[i][j] << " ";
		}
		cout << endl;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t calculatePrimes(int *c, int *a, unsigned int size){
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	primeKernel<<<1, size>>>(dev_c, dev_a);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "primekernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching primekernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	
	return cudaStatus;
}


hipError_t transposeMatrix(int (&matrix_a)[N][M], int(&t_pose)[N][M]) {
	
	hipError_t status;

	int *dev_matrix;
	int *dev_transpose;
	size_t size = (N * M) * sizeof(int);

	status = hipSetDevice(0);
	if(status != hipSuccess) {
		fprintf(stderr,"Failed to set device\n");
	}

	status = hipMalloc((void **) &dev_matrix, size);
	if(status != hipSuccess) {
		fprintf(stderr, "Faield to allocate memory for dev_matrix\n");
	}

	status = hipMemcpy(dev_matrix, matrix_a, size, hipMemcpyHostToDevice);
	if(status != hipSuccess) {
		fprintf(stderr, "Failed to copy matrix_a to dev_matrix and device\n");
		goto Error;
	}

	status = hipMalloc((void **) &dev_transpose, size);
	if(status != hipSuccess) {
		fprintf(stderr, "Faield to allocate memory for dev_matrix\n");
	}

	status = hipMemcpy(dev_transpose, t_pose, size, hipMemcpyHostToDevice);
	if(status != hipSuccess) {
		fprintf(stderr, "Failed to copy t_pose to dev_transpose and device\n");
		goto Error;
	}

	transposeKernal << <N, M >> > (dev_matrix, dev_transpose);

	status = hipGetLastError();
	if(status != hipSuccess) {
		fprintf(stderr, "Failed to %s\n", hipGetErrorString(status));
		goto Error;
	}

	status = hipDeviceSynchronize();
	if(status != hipSuccess) {
		fprintf(stderr, "Failed to sync device %s\n", hipGetErrorString(status));
		goto Error;
	}

	status = hipMemcpy(t_pose, dev_transpose, size, hipMemcpyDeviceToHost);
	if(status != hipSuccess) {
		fprintf(stderr, "Failed to copy dev_transpose to t_pose %s\n", hipGetErrorString(status));
	}



Error:
	hipFree(dev_matrix);
	hipFree(dev_transpose);

	return status;
}
