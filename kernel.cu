#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#define N 10
#define M 10

using namespace std;


hipError_t transposeMatrix(int (&matrix_a)[N][M], int (&t_pose)[N][M]);

__global__ void transposeKernal(int *matrix_a, int *transpose) {
	int i = threadIdx.x;
	int x = blockDim.x;
	int j = blockIdx.x;

	transpose[x * j + i] = matrix_a[x * i + j];
}


int main(){

	size_t size = (N * M) * sizeof(int);
	hipError_t mainStatus;

	int matrix_a[N][M];
	int matrix_b[N][M];
	int matrix_c[N][M];	
	
	for(int i = 0; i < N; i++) {
		for(int j = 0; j < M; j++) {
			matrix_a[i][j] = 1 + rand() / 100;
		}
	}

	mainStatus = transposeMatrix(matrix_a, matrix_b);
	if(mainStatus != hipSuccess) {
		fprintf(stderr, "failed in main\n");
	}

	cout << "Before Transpose" << endl;
	for(int i = 0; i < N; i++) {
		for(int j = 0; j < M; j++) {
			cout << matrix_a[i][j] << " ";
		}
		cout << endl;
	}


	cout << "After Transpose" << endl;
	for(int i = 0; i < N; i++) {
		for(int j = 0; j < M; j++) {
			cout << matrix_b[i][j] << " ";
		}
		cout << endl;
	}

	return 0;
}

hipError_t transposeMatrix(int (&matrix_a)[N][M], int(&t_pose)[N][M]) {
	
	hipError_t status;

	int *dev_matrix;
	int *dev_transpose;
	size_t size = (N * M) * sizeof(int);

	status = hipSetDevice(0);
	if(status != hipSuccess) {
		fprintf(stderr,"Failed to set device\n");
	}

	status = hipMalloc((void **) &dev_matrix, size);
	if(status != hipSuccess) {
		fprintf(stderr, "Faield to allocate memory for dev_matrix\n");
	}

	status = hipMemcpy(dev_matrix, matrix_a, size, hipMemcpyHostToDevice);
	if(status != hipSuccess) {
		fprintf(stderr, "Failed to copy matrix_a to dev_matrix and device\n");
		goto Error;
	}

	status = hipMalloc((void **) &dev_transpose, size);
	if(status != hipSuccess) {
		fprintf(stderr, "Faield to allocate memory for dev_matrix\n");
	}

	status = hipMemcpy(dev_transpose, t_pose, size, hipMemcpyHostToDevice);
	if(status != hipSuccess) {
		fprintf(stderr, "Failed to copy t_pose to dev_transpose and device\n");
		goto Error;
	}

	transposeKernal << <N, M >> > (dev_matrix, dev_transpose);

	status = hipGetLastError();
	if(status != hipSuccess) {
		fprintf(stderr, "Failed to %s\n", hipGetErrorString(status));
		goto Error;
	}

	status = hipDeviceSynchronize();
	if(status != hipSuccess) {
		fprintf(stderr, "Failed to sync device %s\n", hipGetErrorString(status));
		goto Error;
	}

	status = hipMemcpy(t_pose, dev_transpose, size, hipMemcpyDeviceToHost);
	if(status != hipSuccess) {
		fprintf(stderr, "Failed to copy dev_transpose to t_pose %s\n", hipGetErrorString(status));
	}



Error:
	hipFree(dev_matrix);
	hipFree(dev_transpose);

	return status;
}
